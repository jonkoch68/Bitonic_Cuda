#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>
#include <vector>
#include <chrono>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <algorithm>

class Timer {
public:
	std::chrono::system_clock::time_point Begin;
	std::chrono::system_clock::time_point End;
	std::chrono::system_clock::duration RunTime;
	Timer() {//constructor
		Begin = std::chrono::system_clock::now();
	}
	~Timer() {
		End = std::chrono::system_clock::now();
		RunTime = End - Begin;
		printf("%llu us\n", std::chrono::duration_cast<std::chrono::microseconds>(RunTime).count());
	}
};

static int* Data = 0;

__device__ void cudaSwap(int* a, int* b)
{
	int temp = *a;
	*a = *b;
	*b = temp;
}

__global__ void SortKernel(int* array, uint64_t size, uint64_t originalSize)
{
	uint64_t threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
	uint64_t step = size / 2;
	uint64_t startIdx = ((threadId / step) * size) + (threadId % step);
	uint64_t endIdx = startIdx + step;
	// false is down, true is up
	bool direction = ((threadId / (originalSize / 2)) % 2) == 0 ? false : true;

	if (direction) //If swapping upwards
	{
		if (array[startIdx] < array[endIdx])
		{
			cudaSwap(&array[startIdx], &array[endIdx]);
		}
	}
	else
	{
		if (array[startIdx] > array[endIdx])
		{
			cudaSwap(&array[startIdx], &array[endIdx]);
		}
	}
}

__global__ void SortKernelShared(int* array, uint64_t size, uint64_t originalSize, int numThreads)
{
	uint64_t GlobalThreadId = (blockIdx.x * blockDim.x) + threadIdx.x;
	uint64_t step = size / 2;

	// false is down, true is up
	bool direction = ((GlobalThreadId / (originalSize / 2)) % 2) == 0 ? false : true;
	//Range of numbers each thread will be responsible for adding 
	uint64_t range = (size/2)  / numThreads;
	__shared__ int sharedArr[2048]; // 32,768 Bytes. Cannot exceed this without going over 48k

	//Load memory into shared L1 cache from global
	int localIdx = threadIdx.x;
	uint64_t globalIdx = GlobalThreadId;
	int i = 0;
	do
	{
		// Get Indexes
		int startIdx = ((localIdx / step) * size) + (localIdx % step);
		uint64_t GlobalStartIdx = ((globalIdx / step) * size) + (globalIdx % step);
		//Put global data into shared memory
		sharedArr[startIdx] = array[GlobalStartIdx];
		sharedArr[startIdx + step] = array[GlobalStartIdx + step];
		//Increment by number of threads
		localIdx += numThreads;
		globalIdx += numThreads;
		//Increment i by range
		i++;
	} while (i < range);

	__syncthreads(); //Wait for all threads to finish loading from global memory

	//Begin operation 
	int stageSize = size;
	int stageStep = stageSize/2;
	int endIdx, startIdx;
	localIdx = threadIdx.x;
	int adjustedRange = range;
	while (stageSize != 1)
	{
		i = 0;
		do
		{
			startIdx = ((localIdx / stageStep) * stageSize) + (localIdx % stageStep);
			endIdx = startIdx + stageStep;
			if (endIdx >= 8192) break;
			if (direction) //If swapping upwards
			{
				if (sharedArr[startIdx] < sharedArr[endIdx])
				{
					cudaSwap(&sharedArr[startIdx], &sharedArr[endIdx]);
				}
			}
			else
			{
				if (sharedArr[startIdx] > sharedArr[endIdx])
				{
					cudaSwap(&sharedArr[startIdx], &sharedArr[endIdx]);
				}

			}
			localIdx += numThreads;
			i++;
		} while (i < adjustedRange);

		//Wait for all threads to finished before continuing to the next level
		__syncthreads();
		stageSize = stageStep;
		stageStep = stageSize / 2;
		//adjustedRange *= 2;
		localIdx = threadIdx.x;
	}

	localIdx = threadIdx.x;
	globalIdx = GlobalThreadId;
	i = 0;
	do
	{
		// Get Indexes
		startIdx = ((localIdx / step) * size) + (localIdx % step);
		uint64_t GlobalStartIdx = ((globalIdx / step) * size) + (globalIdx % step);
		//Put global data into shared memory
		array[GlobalStartIdx] = sharedArr[startIdx];
		array[GlobalStartIdx + step] = sharedArr[startIdx + step];
		//Increment by number of threads
		localIdx += numThreads;
		globalIdx += numThreads;
		//Increment i by range
		i++;
	} while (i < range);

	__syncthreads();
}

void SharedBitonic(int* aData, uint64_t aSize)
{
	hipError_t cudaStatus;
	int* dev_array = 0;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return ;
	}

	//Allocate memory on device (GPU)
	cudaStatus = hipMalloc((void**)&dev_array, aSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_array);
		return ;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_array, aData, aSize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_array);
		return ;
	}

	// Get Device info
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	std::printf("Device Number: %d\n", 0);
	printf("  Device name: %s\n", prop.name);
	printf("  Memory Clock Rate (KHz): %d\n",
		prop.memoryClockRate);
	printf("  Memory Bus Width (bits): %d\n",
		prop.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %f\n",
		2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	printf("Shared Memory per Multiprocessor %d\n", prop.sharedMemPerMultiprocessor);
	int maxThreads = prop.maxThreadsPerBlock;
	int maxBlocks = ((aSize / 2) + maxThreads - 1) / maxThreads;
	if (maxThreads > aSize / 2)
	{
		maxBlocks = 1;
		maxThreads = aSize / 2;
	}
	printf("  Max Threads per block: %d\n  Max Blocks: %d\n", maxThreads, maxBlocks);

	dim3 threads(maxThreads, 1);
	dim3 blocks(maxBlocks, 1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernelShared), hipFuncCachePreferL1);
	{
		Timer T;
		for (uint64_t i = 2; i <= aSize; i *= 2)
		{
			//printf("next iter\n");
			uint64_t j = i;
			while (j != 1)
			{
				// 48Kb of shared memory / 12,000 int32 values stored. 
				// 8,192 entries shared to without going over 48k in shared memory
				// 4,096 
				if (j <= std::pow(2, 11)) 
				{

					SortKernelShared << <blocks, threads >> > (dev_array, j, i, maxThreads);
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "SortKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
						hipFree(dev_array);
						return;
					}
					break;
				}
				else
				{
					//printf("size: %d \n", j);
					SortKernel << <blocks, threads >> > (dev_array, j, i);
					// Check for any errors launching the kernel
					cudaStatus = hipGetLastError();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "SortKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
						hipFree(dev_array);
						return;
					}
				}
				j = j / 2;
			}
			// REMEMBER TO REMOVE
			cudaStatus = hipMemcpy(aData, dev_array, aSize * sizeof(int), hipMemcpyDeviceToHost);
		}
		printf("Runtime took ");
	}
	printf("Processing Completed\n");
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SortKernel!\n", cudaStatus);
		hipFree(dev_array);
		return;
	}
	printf("Sync Completed\n");
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(aData, dev_array, aSize * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_array);
		return;
	}
}


int main()
{
	printf("Running Shared Optimized\n");


	// Declare range of random numbers
	const int range = 10;

	//Declare problem size
	const uint64_t size = pow(2, 16);
	printf("Sorting %llu values\n", size);
	// Allocate memory on host device
	Data = (int*)std::malloc(size * sizeof(int));
	printf("Allocating %llu bytes of memory\n", size * sizeof(int));
	// randomly fill array
	for (uint64_t i = 0; i < size; i++)
	{
		Data[i] = rand() % range;
	}

	{
		Timer T;
		SharedBitonic(Data, size);
		printf("Execution time: ");
	}

	printf("Transfer Completed\n");
	printf("confirming solition\n");
	int prev = 0;
	bool passed = true;
	for (uint64_t i = 0; i < size; i++)
	{
		if (Data[i] < prev)
		{
			//printf("failed at %llu Current %llu, Prev %llu\n", i, Data[i], prev);
			passed = false;
		}
		prev = Data[i];
	}

	if (passed) printf("All %llu number correctly sorted\n", size);
	else printf("Failed, incorrect sorting value\n");
	
	printf("Printing results\n");
	std::ofstream out1("Output1.txt");
	prev = 0;
	for (int i = 0; i < size; i++)
	{
		out1 << Data[i] << " ";
		if (Data[i] < prev ) out1 << "\n\n\n";
		prev = Data[i];
	}
	
}